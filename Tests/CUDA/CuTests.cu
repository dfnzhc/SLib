#include "hip/hip_runtime.h"
﻿/**
 * @File cuTests.cu
 * @Author dfnzhc (https://github.com/dfnzhc)
 * @Date 2025/7/3
 * @Brief This file is part of SLib.
 */

#include "CuTests.hpp"
#include "SLibCUDA/Math.hpp"

#define T(expr) ((expr) ? 0 : 1)

__global__ void TestAbs(int* numErrors)
{
    // numErrors += T(slib::Abs(-42) == 42);
}

int TestCommon()
{
    int* dev_pNumErrors = nullptr;

    hipMalloc(&dev_pNumErrors, sizeof(int));
    
    TestAbs<<<1, 1>>>(dev_pNumErrors);

    int numErrors;

    hipMemcpy(&numErrors, dev_pNumErrors, sizeof(int), hipMemcpyDeviceToHost);

    return numErrors;
}